#include "hip/hip_runtime.h"
// CUDA Dependencies
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// GL Dependencies
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include "data/kernel/reduction.h"
#include "data/kernel/reduction_device.h"
#include "utils/sharedmemory.h"
#include "utils/cuda_common.h"

/*
 * Various kernels for reduction.
 * See https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
 */
template <unsigned int block_size, typename T, class Op>
__inline__ __device__
T block_reduce_1(T* input)
{
  extern __shared__ T shared_data[block_size];
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x + thread;
  shared_data[thread] = input[index];
  __syncthreads();

  // I think blockDim.x should be equal to block_size
  for(unsigned int s = 1; s < blockDim.x; ++s)
  {
    if(thread % (2*s) == 0)
    {

      shared_data[thread] = Op::op(shared_data[thread], shared_data[thread + s]);
    }
    __syncthreads();
  }
  if (thread == 0)
  {
    return shared_data[thread];
  }
}

/*
 * Interleaved addressing, which leads to bank conflict
 */
template <unsigned int block_size, typename T, class Op>
__inline__ __device__
T block_reduce_2(T* input)
{
  extern __shared__ T shared_data[block_size];
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x + thread;
  shared_data[thread] = input[index];
  __syncthreads();

  for (unsigned int s = 1; s < block_size; ++s)
  {
    int current = 2 * thread * s;
    if (current + s < block_size)
    {
      shared_data[current] = Op::op(shared_data[thread], shared_data[thread + s]);
    }
    __syncthreads();
  }
  if (thread == 0)
  {
    return shared_data[thread];
  }
}

/*
 * Sequential addressing
 */
template <unsigned int block_size, typename T, class Op>
__inline__ __device__
T block_reduce_3(T* input)
{
  extern __shared__ T shared_data[block_size];
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x + thread;
  shared_data[thread] = input[index];
  __syncthreads();

  // Half of the threads are idle on the first loop iteration
  for (unsigned int s = block_size / 2; s > 0; s >>= 1)
  {
   if (thread < s)
   {
     shared_data[thread] = Op::op(shared_data[thread], shared_data[thread + s]);
   }
    __syncthreads();
  }
  if (thread == 0)
  {
    return shared_data[thread];
  }
}

/*
 * Sequential addressing and first reduction outside the loop
 * Halve block_size (== blockDim.x) in the template
 * TODO: why do we even need this first template parameter?
 */
template <unsigned int halved_block_size, typename T, class Op>
__inline__ __device__
T block_reduce_4(T* input)
{
  extern __shared__ T shared_data[halved_block_size];
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x * 2 + thread;
  shared_data[thread] = input[index] + input[index + halved_block_size];
  __syncthreads();

  // Half of the threads are idle on the first loop iteration
  for (unsigned int s = halved_block_size; s > 0; s >>= 1)
  {
   if (thread < s)
   {
     shared_data[thread] = Op::op(shared_data[thread], shared_data[thread + s]);
   }
    __syncthreads();
  }

  if (thread == 0)
  {
    return shared_data[thread];
  }
}

/*
 * When s <= 32, only one warp (unit for SIMD) exists.
 * Unroll the last six loop by defining "warp_reduce".
 * Use the volatile keyword, according to
 * https://stackoverflow.com/questions/15331009/when-to-use-volatile-with-shared-cuda-memory
 */
template <typename T>
__device__
void warp_reduce(volatile T* shared_data, int thread)
{
  shared_data[thread] += shared_data[thread + 32];
  shared_data[thread] += shared_data[thread + 16];
  shared_data[thread] += shared_data[thread +  8];
  shared_data[thread] += shared_data[thread +  4];
  shared_data[thread] += shared_data[thread +  2];
  shared_data[thread] += shared_data[thread +  1];
}

// TODO: the return type is incorrect
template <unsigned int halved_block_size, typename T, class Op>
__inline__ __device__
T block_reduce_5(T* input)
{
  extern __shared__ T shared_data[halved_block_size];
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x * 2 + thread;
  shared_data[thread] = input[index] + input[index + halved_block_size];
  __syncthreads();

  for (unsigned int s = halved_block_size; s > 32; s >>= 1)
  {
   if (thread < s)
   {
     shared_data[thread] = Op::op(shared_data[thread], shared_data[thread + s]);
   }
    __syncthreads();
  }

  if (thread < 32)
  {
    warp_reduce(shared_data, thread);
  }
  __syncthreads();

  if (thread == 0)
  {
    return shared_data[thread];
  }
}

template <typename T, class Op>
__global__
void segmented_reduce_1(T *val,
                         unsigned int num_val,
                         int* keys,
                         unsigned int num_keys,
                         int* key_ranges,
                         T* output)
{
  __shared__ int s_key[256];
  __shared__ int s_val[256];
  T* s_arr = SharedMemory<T>();

  int min_key = key_ranges[2 * blockIdx.x];
  int key_diff = key_ranges[2 * blockIdx.x + 1] - min_key;
  unsigned int thread = threadIdx.x;
  unsigned int index = blockIdx.x * blockDim.x + thread;

  s_val[thread] = val[thread];
  s_key[thread] = keys[thread];
  if(thread <= key_diff)
  {
    s_arr[thread] = 0;
  }

  __syncthreads();

  for(int i = 1; i < blockDim.x; i *= 2)
  {
    if(thread % (i * 2) == 0) // Branch divergence!
    {
      int w0 = s_key[thread];
      int w1 = s_key[thread + i];

      if (w0 != w1) // If keys are different
      {
        s_arr[w1 - min_key] += s_val[thread + i];
      }
      else
      {
        s_val[thread] += s_val[thread + i];
      }
    }
    __syncthreads();

    if (thread <= key_diff)
    {
      atomicAdd(&output[min_key + thread], s_arr[thread]);
    }
    __syncthreads();
    if(thread == 0)
    {
      atomicAdd(&output[s_key[0]], s_val[0]);
    }
  }

}

// Key preprocessing for segmented reduction
__global__
void get_keyrange_per_block(int* keys,
                            int* ranges,
                            int n, int block_size)
{
  int thread = threadIdx.x;
  if(thread < block_size)
  {
    ranges[2*thread] = keys[block_size * thread];
    ranges[2*thread + 1] = keys[block_size * (thread+1) - 1];
  }
}

/*
 *
 */
template <typename T, class Op>
void segmented_reduce(T *val,
                      unsigned int num_val,
                      int* keys,
                      unsigned int num_keys,
                      T* output)
{
  unsigned int num_threads = 256;
  unsigned int num_blocks = num_val / num_threads; // TODO: Remove the constraint on num_val
  int* keyranges;
  hipMalloc((void**) *keyranges, 2 * num_blocks * sizeof (int));
  checkCudaError("hipMalloc failure");

  dim3 dim_block_ranges(num_blocks, 1, 1);
  dim3 dim_grid_ranges(1, 1, 1);
  dim3 dim_block(num_threads, 1, 1);
  dim3 dim_grid(num_blocks, 1, 1);
  unsigned int sme_size = num_threads * 3 * sizeof (int); // What does this mean?
  segmented_reduce_1<T, Op><<<dim_grid, dim_block, sme_size>>>(val, num_val, keys, num_keys, keyranges, output);
  checkCudaError("Segmented reduction kernel call failed");
  hipFree(keyranges);
}

template<typename T, class ReductionOp, unsigned int block_size>
__inline__ __device__
T reduction_device(T* arr)
{
  unsigned int thread = threadIdx.x;
  if(block_size >= 512)
  {
    if(thread < 256)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 256]);
    }
    __syncthreads();
  }

  if(block_size >= 256)
  {
    if(thread < 128)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 128]);
    }
  }
  __syncthreads();

  if(block_size >= 128)
  {
    if(thread < 64)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 64]);
    }
  }
  __syncthreads();

  if(block_size >= 64)
  {
    if(thread < 32)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 32]);
    }
  }
  __syncthreads();

  if(block_size >= 32)
  {
    if(thread < 16)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 16]);
    }
  }
  __syncthreads();

  if(block_size >= 16)
  {
    if(thread < 8)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 8]);
    }
  }
  __syncthreads();

  if(block_size >= 8)
  {
    if(thread < 4)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 4]);
    }
  }
  __syncthreads();

  if(block_size >= 4)
  {
    if(thread < 2)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 2]);
    }
  }
  __syncthreads();

  if(block_size >= 2)
  {
    if(thread < 1)
    {
      arr[thread] = ReductionOp::op(arr[thread], arr[thread + 1]);
    }
  }
  __syncthreads();

  return arr[0];
}

/*
 * This is basically a block reduction, since we are placing
 */
template <typename T, class Op, unsigned int chunk_size>
__global__
void chunk_reduce_global(T* val,
                     int* starting_indices,
                     int* chunk_len,
                     T* output)
{
  T* s_arr = SharedMemory<T>();
  unsigned int thread = threadIdx.x;
  unsigned int chunk_index = blockIdx.x;
  __shared__ unsigned int starting_index;
  __shared__ unsigned int len;

  if (thread == 0)
  {
    starting_index = starting_indices[chunk_index];
    len = chunk_len[chunk_index];
  }
  __syncthreads();

  if (thread < len) // Branching?
  {
    s_arr[thread] = val[starting_index + thread];
  }
  else
  {
    s_arr[thread] = Op::identity();
  }

  __syncthreads();

  // reduce
  T res = reduction_device<T, Op, chunk_size>(s_arr);

  if (thread == 0)
  {
    output[chunk_index] = res;
  }
}

template <typename T, class Op, unsigned int chunk_size>
void chunk_reduce(T* val, int* starting_indices,
                  int* chunk_len, unsigned int num_chunks,
                  T* output)
{
  dim3 block_dim = dim3(chunk_size, 1, 1);
  dim3 grid_dim = dim3(num_chunks, 1, 1);
  int size = chunk_size * sizeof (T);
  chunk_reduce<T, Op, chunk_size> <<<block_dim, grid_dim>>>(val, starting_indices, chunk_len, output);
}
